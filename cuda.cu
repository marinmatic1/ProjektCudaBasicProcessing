#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "loading.h"
#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"
#include<cmath>

const int xBlockSize = 32;
const int yBlockSize = 32;

typedef struct {
	int x, y;
	float *grey;

} d_image;
void mat_u_niz(PGMImage izvorna, float *pomIzv)
{
	for (int i = 0; i<izvorna.y; i++)
		for (int j = 0; j<izvorna.x; j++)
			pomIzv[i*izvorna.x + j] = izvorna.grey[i][j];
}

__global__ void NekaFunkcija(d_image d_izvorna, float *nizS)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int num = d_izvorna.x;
	float Kernel[3][3] = {
		{ 0.07513,0.12384,0.07513 },
		{ 0.12384,0.20418,0.12384 },
		{ 0.07513,0.12384,0.07513 }
	};

	float sum;
	if (row<(d_izvorna.y - 1) && (col<d_izvorna.x - 1))
	{
		sum = 0.0;
		for (int k = -1; k <= 1; k++)
		{
			for (int j = -1; j <= 1; j++)
			{
				sum = sum + Kernel[j + 1][k + 1] * (d_izvorna.grey[(row - j)*num + (col - k)]);
			}
		}
		nizS[row*num + col] = sum;
	}
}


int main()
{
	GpuTimer timer;
	PGMImage izvorna, nova;

	d_image d_izvorna;
	float *d_nova;
	float *izvorniNiz, *pom;
	hipError_t err;
	char file1[1024] = "fpmoz01.pgm";
	char file2[1024] = "fpmoz10.pgm";

	ucitajPGM(file1, &izvorna);

	alloc_matrix(&nova.grey, izvorna.y, izvorna.x);
	nova.x = izvorna.x;
	nova.y = izvorna.y;

	//alociramo memoriju za pomocni niz i izvorni niz
	pom = (float *)malloc(nova.x*nova.y*sizeof(float));
	izvorniNiz = (float *)malloc(nova.x*nova.y*sizeof(float));

	//prebacujemo izvornu matricu(sliku) u niz 
	mat_u_niz(izvorna, izvorniNiz);

	d_izvorna.x = izvorna.x;
	d_izvorna.y = izvorna.y;

	// veličinu memorije u bajtovima za niz
	int size = d_izvorna.x * d_izvorna.y * sizeof(float);
	//alociranje memorije naGPU
	hipMalloc((void **)& d_izvorna.grey, size);
	hipMalloc((void **)& d_nova, size);

	//kopiranje na GPU
	hipMemcpy(d_izvorna.grey, izvorniNiz, size, hipMemcpyHostToDevice);
	//u blkSize funkciji imamo broj dretvi po bloku
	dim3 blkSize(xBlockSize, yBlockSize);
	//numBlock nam pokazuje koliko blokova imamo
	dim3 numBlock(ceil((float)izvorna.x / xBlockSize), ceil((float)izvorna.y / yBlockSize));

	//pocetak mjerenja
	timer.Start();
	//jezgrena funkcija
	NekaFunkcija << < numBlock, blkSize >> > (d_izvorna, d_nova);
	err = hipDeviceSynchronize();
	//zavrsetak mjerenja
	timer.Stop();

	//ceka zavrsetak svih radnji za nastavak

	printf("Izvedba kernela: %s\n", hipGetErrorString(err));

	printf("Vrijeme izvrsenja u cudi je = %g ms\n", timer.Elapsed());

	//kopiranje sa GPU na host
	err = hipMemcpy(pom, d_nova, size, hipMemcpyDeviceToHost);
	printf("Kopiranje na host: %s\n", hipGetErrorString(err));

	//prebacivanje niza u matricu
	for (int i = 0; i < nova.y; i++)
		for (int j = 0; j < nova.x; j++)
			nova.grey[i][j] = pom[i*nova.x + j];

	zapisiPGM(file2, &nova);

	//oslobađanje memorije
	hipFree(d_izvorna.grey);
	hipFree(d_nova);
	free(pom);
	free(izvorniNiz);

	disalloc_matrix(izvorna.grey, izvorna.y, izvorna.x);
	disalloc_matrix(nova.grey, nova.y, nova.x);

	printf("Press any key...");
	getchar();
}
